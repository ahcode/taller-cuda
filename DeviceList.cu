#include <hip/hip_runtime.h>

#include <stdio.h>

int main(){
	int deviceCount; hipGetDeviceCount(&deviceCount);
	int device;
	for (device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("Device %d (%s) has compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
}
