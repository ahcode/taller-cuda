#include <hip/hip_runtime.h>

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <sys/time.h>

//Tamaño de matrices (cuadradas)
#define N 1024

//Kernel
__global__ void mul(int * A, int * B, int * C){
	int i = blockIdx.x;
	int j = threadIdx.x;
	__shared__ int aux[N];
	aux[j] = 0;
	for (int k = 0; k < N; k++){
		aux[j] += A[i * N + k] * B[k * N + j];
	}
	__syncthreads();
	C[i * N + j] = aux[j];
}


int main(){
	struct timeval t1, t2;
	int *hA, *hB, *hC, *hC2; //Host Matrix
	int *dA, *dB, *dC; //Device Matrix

	//Reserva de memoria Host
	hA = (int*)malloc(N*N*sizeof(int));
	hB = (int*)malloc(N*N*sizeof(int));
	hC = (int*)malloc(N*N*sizeof(int));
	hC2 = (int*)malloc(N*N*sizeof(int));

	//Inicialización de matrices
	srand(time(NULL));
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			hA[i*N+j] = rand();
			hB[i*N+j] = rand();
		}
	}

	//Reserva de memoria GPU
	hipMalloc((void **)&dA, N*N*sizeof(int));
	hipMalloc((void **)&dB, N*N*sizeof(int));
	hipMalloc((void **)&dC, N*N*sizeof(int));

	//Copia Host -> GPU
	hipMemcpy(dA, hA, N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, N*N*sizeof(int), hipMemcpyHostToDevice);

	gettimeofday(&t1, 0);

	//Ejecución Kernel
	mul<<<N, N>>>(dA, dB, dC);

	hipDeviceSynchronize();
	gettimeofday(&t2, 0);

	//Copia Device -> Host
	hipMemcpy(hC, dC, N*N*sizeof(int), hipMemcpyDeviceToHost);

	//Multiplicación en Host
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			hC2[i*N + j] = 0;
			for(int k = 0; k < N; k++){
				hC2[i*N + j] += hA[i*N + k] * hB[k*N + j];
			}
		}
	}

	//Comprobación de errores
	bool error = false;
	for(int i = 0; i < N*N; i++){
		if(hC[i] != hC2[i]){
			error = true;
			break;
		}
	}

	if(error)
		printf("La multiplicación de matrices ha fallado.\n");
	else
		printf("Multiplicación de matrices correcta.\n");

	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
	printf("Tiempo: %f ms\n", time);

	//Liberar memoria
	free(hA);
	free(hB);
	free(hC);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}
